#include "hip/hip_runtime.h"
// Copyright (c) 2011 Kerem KAT 
// 
// http://dissipatedheat.com/
// Do not hesisate to contact me about usage of the code or to make comments 
// about the code. Your feedback will be appreciated.
// keremkat<@>gmail<.>com
//
// Kodun kullan�m� hakk�nda veya yorum yapmak i�in benimle ileti�im kurmaktan
// �ekinmeyiniz. Geri bildirimleriniz de�erlendirilecektir.
// keremkat<@>gmail<.>com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to 
// deal in the Software without restriction, including without limitation the 
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software is 
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

/**
	\file texBoxBlur.cu
	CUDA texture box blur kernelinin launcher metodunu ve kernelini tan�mlar.
*/

#include "texBoxBlur.h"

texture<float4, 2, hipReadModeElementType> texBlur1; /**< Kernelde kullan�lan texture sembol�. */

#define BLOCK_SIZE_X (32) /**< Yatay blok boyutu */
#define BLOCK_SIZE_Y (32) /**< D��ey blok boyutu */

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 1

/**	
	Texture kullanarak g�r�nt�n�n 5x5 box blurunu alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/
__global__
void gpuTexBoxBlur(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	float tw = 1.0f / width; // Texture kordinatlar�nda 1 pixel geni�lik.
	float th = 1.0f / height; // Texture kordinatlar�nda 1 pixel y�kseklik.

	float tu = ( float )col * tw;
	float tv = ( float )row * th;


	float4 texVal = make_float4(0, 0, 0, 0);

	tw *= 8;
	th *= 8;

#pragma unroll
	for(int i = 0; i < 5; i++)
	{
#pragma unroll
		for(int j = 0; j < 5; j++)
		{			
			texVal += tex2D( texBlur1, tu + ( i - 2 ) * tw, tv + ( j - 2 ) * th );
		}
	}

	texVal *= 1.0f / 25;


	*( image + cIdx )     = texVal.x;
	*( image + cIdx + 1 ) = texVal.y;
	*( image + cIdx + 2 ) = texVal.z;
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTexBoxBlur kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTexBoxBlurLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE_X, BLOCK_SIZE_Y );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexBoxBlur<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuTexBoxBlur kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif