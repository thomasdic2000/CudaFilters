#include "hip/hip_runtime.h"
// Copyright (c) 2011 Kerem KAT 
// 
// http://dissipatedheat.com/
// Do not hesisate to contact me about usage of the code or to make comments 
// about the code. Your feedback will be appreciated.
// keremkat<@>gmail<.>com
//
// Kodun kullan�m� hakk�nda veya yorum yapmak i�in benimle ileti�im kurmaktan
// �ekinmeyiniz. Geri bildirimleriniz de�erlendirilecektir.
// keremkat<@>gmail<.>com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to 
// deal in the Software without restriction, including without limitation the 
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software is 
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

/**
	\file texInvert.cu
	CUDA texture invert kernelinin launcher metodunu ve kernelini tan�mlar.
*/

#include "texInvert.h"

texture<float4, 2, hipReadModeElementType> texInvert1; /**< Kernelde kullan�lan texture sembol�. */

#define BLOCK_SIZE (32) /**< Blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare blok ). */

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 0

/**	
	Texture kullanarak g�r�nt�n�n negatifini alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.
*/
__global__
void gpuTexInvert(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.


	// threade ait kordinat�n texture uzay�ndaki kordinat� bulunur.
	float tu = (float)col / width;
	float tv = (float)row / height;

	// Texture �zerinden g�r�nt� verisi okunur.
	float4 texVal = tex2D( texInvert1, tu, tv );

	// Texture de�erleri 1'den ��kart�larak global belle�e yaz�l�r.
	*( image + cIdx )     = 1 - texVal.x;
	*( image + cIdx + 1 ) = 1 - texVal.y;
	*( image + cIdx + 2 ) = 1 - texVal.z;
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTexInvert kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTexInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif