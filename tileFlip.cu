#include "hip/hip_runtime.h"
// Copyright (c) 2011 Kerem KAT 
// 
// http://dissipatedheat.com/
// Do not hesisate to contact me about usage of the code or to make comments 
// about the code. Your feedback will be appreciated.
// keremkat<@>gmail<.>com
//
// Kodun kullan�m� hakk�nda veya yorum yapmak i�in benimle ileti�im kurmaktan
// �ekinmeyiniz. Geri bildirimleriniz de�erlendirilecektir.
// keremkat<@>gmail<.>com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to 
// deal in the Software without restriction, including without limitation the 
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software is 
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include "tileFlip.h"

/**
	\file tileFlip.cu
	CUDA tile flip kernelinin launcher metodunu ve kernelini tan�mlar.
*/

/** Kernel 1 griddeki blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare bloklar ). */
#define BLOCK_SIZE (32)

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 1

/**	
	G�r�nt�y� blok blok �eviren kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/__global__
void gpuTileFlip(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y; // sat�r No.

	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x; // s�tun No.

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	/*
	       *( image + linearIndex ): Blue, in [0, 1]
		   *( image + linearIndex + 1 ): Green, in [0, 1]
		   *( image + linearIndex + 2 ): Red, in [0, 1]
	*/

	__shared__ float smBlockB[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float smBlockG[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float smBlockR[BLOCK_SIZE][BLOCK_SIZE];

	smBlockB[threadIdx.x][threadIdx.y] = image[ cIdx ];
	smBlockG[threadIdx.x][threadIdx.y] = image[ cIdx + 1 ];
	smBlockR[threadIdx.x][threadIdx.y] = image[ cIdx + 2 ];

	__syncthreads();	
	
	image[ cIdx ]     =	smBlockB[threadIdx.y][threadIdx.x];
	image[ cIdx + 1 ] = smBlockG[threadIdx.y][threadIdx.x];
	image[ cIdx + 2 ] = smBlockR[threadIdx.y][threadIdx.x];
		
	
	//image[ cIdxRight + 2 ] = 0;

	/**( image + cIdx ) = abs((*( image + cIdx ) - *( image + cIdxRight )));
	*( image + cIdx + 1 ) = abs((*( image + cIdx + 1 ) - *( image + cIdxRight + 1 )));
	*( image + cIdx + 2 ) = abs((*( image + cIdx + 2 ) - *( image + cIdxRight + 2 )));*/
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTileFlip kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTileFlipLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif
	
    gpuTileFlip<<< dimGrid, dimBlock >>>( d_Image, width, height);
	
#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
