#include "hip/hip_runtime.h"
// Copyright (c) 2011 Kerem KAT 
// 
// http://dissipatedheat.com/
// Do not hesisate to contact me about usage of the code or to make comments 
// about the code. Your feedback will be appreciated.
// keremkat<@>gmail<.>com
//
// Kodun kullan�m� hakk�nda veya yorum yapmak i�in benimle ileti�im kurmaktan
// �ekinmeyiniz. Geri bildirimleriniz de�erlendirilecektir.
// keremkat<@>gmail<.>com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to 
// deal in the Software without restriction, including without limitation the 
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software is 
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#ifndef TEX_INVERT_CU
#define TEX_INVERT_CU

/**
	\file texAbsDiff.cu
	CUDA texture absolute difference kernelinin launcher metodunu ve kernelini tan�mlar.
*/

#include "texAbsDiff.h"

texture<float4, 2, hipReadModeElementType> texAbsDiff1; /**< Kernelde kullan�lan texture sembol�. */

#define BLOCK_SIZE_X (32) /**< Yatay blok boyutu */
#define BLOCK_SIZE_Y (32) /**< D��ey blok boyutu */

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 1


/**
	Mutlak de�er metodu.

	\param a Mutlak de�eri al�nacak float4 de�eri.

	a parametresinin her bir eleman�n�n mutlak de�erini alarak yeni bir float4 d�nd�r�r.
*/
inline __host__ __device__ float4 abs( float4 a )
{	
	return make_float4( fabsf( a.x ), fabsf( a.y ), fabsf( a.z ), fabsf( a.w ) );
}

/**	
	Texture kullanarak g�r�nt�n�n kom�u pikseller ile mutlak fark�n� alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/
__global__
void gpuTexAbsDiff(
	float* image,
	int width,
	int height
	)
{
	// threade ait sat�r ve s�tunu hesapla.
	int row = blockIdx.y * BLOCK_SIZE_Y + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE_X + threadIdx.x;

	// global memorydeki imaj i�in indisi hesapla.
	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	float tw = 1.0f / width; // Texture kordinatlar�nda 1 pixel geni�lik.
	float th = 1.0f / height; // Texture kordinatlar�nda 1 pixel y�kseklik.

	// merkez piksel kordinat�n normalize texel kordinatlar�n� hesapla.
	float tu = ( float )col * tw;
	float tv = ( float )row * th;

	/*tw *= 4;
	th *= 4;*/
	
	// row, col etraf�ndaki 4 ana y�ndeki texelden farklar�n� al.
	float4 texVal1 = tex2D( texAbsDiff1, tu, tv ) - tex2D( texAbsDiff1, tu + tw, tv + th );
	float4 texVal2 = tex2D( texAbsDiff1, tu, tv ) - tex2D( texAbsDiff1, tu - tw, tv - th );
	float4 texVal3 = tex2D( texAbsDiff1, tu, tv ) - tex2D( texAbsDiff1, tu + tw, tv - th );
	float4 texVal4 = tex2D( texAbsDiff1, tu, tv ) - tex2D( texAbsDiff1, tu - tw, tv + th );

	// 4 ana y�ndeki farklar�n mutlak de�erlerinin ortalamas�n� al.
	float4 texVal = 0.25f * ( abs( texVal1 ) + abs( texVal2 ) + abs( texVal3 ) + abs( texVal4 ) );

	// global memorydeki imaja hesaplanan de�erleri aktar.
	*( image + cIdx )     = texVal.x;
	*( image + cIdx + 1 ) = texVal.y;
	*( image + cIdx + 2 ) = texVal.z;
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuTexAbsDiff kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceTexAbsDiffLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE_X, BLOCK_SIZE_Y );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuTexAbsDiff<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuTexAbsDiff kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}


#endif