#include "hip/hip_runtime.h"
// Copyright (c) 2011 Kerem KAT 
// 
// http://dissipatedheat.com/
// Do not hesisate to contact me about usage of the code or to make comments 
// about the code. Your feedback will be appreciated.
// keremkat<@>gmail<.>com
//
// Kodun kullan�m� hakk�nda veya yorum yapmak i�in benimle ileti�im kurmaktan
// �ekinmeyiniz. Geri bildirimleriniz de�erlendirilecektir.
// keremkat<@>gmail<.>com
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to 
// deal in the Software without restriction, including without limitation the 
// rights to use, copy, modify, merge, publish, distribute, sublicense, and/or 
// sell copies of the Software, and to permit persons to whom the Software is 
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.

#include "invert.h"

/**
	\file invert.cu
	CUDA invert kernelinin launcher metodunu ve kernelini tan�mlar.
*/

/** Kernel 1 griddeki blok boyutu ( BLOCK_SIZE x BLOCK_SIZE kare bloklar ). */
#define BLOCK_SIZE (32)

/** GPU zaman�n� �l�mek i�in 1 yap�n�z. */
#define ENABLE_TIMING_CODE 0

/**	
	G�r�nt�n�n tersini alan kernel.

	\param image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	
	Metod GPU �zerinde �al���r, ��kt�s�n� image parametresinin �zerine yazar.

	*/
__global__
void gpuInvert(
	float* image,
	int width,
	int height
	)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	int cIdx = ( row * width + col ) * 3; // 3 ile �arp�m RGB i�in, linearIndex.

	// normalize edilmi� pikselleri 1'den ��kartt���m�zda g�r�nt�n�n negatifini alm�� oluruz.
	*( image + cIdx     ) = 1 - *( image + cIdx     ); // Blue kanal�
	*( image + cIdx + 1 ) = 1 - *( image + cIdx + 1 ); // Green kanal�
	*( image + cIdx + 2 ) = 1 - *( image + cIdx + 2 ); // Red kanal�
}

/**
	\ref ptKernelLauncher tipinde metod.

	\param d_Image [0, 1] aral���na normalize edilmi�, BGR kanal s�ral� g�r�nt�n�n GPU belle�indeki adresi.
	\param width G�r�nt�n�n piksel olarak geni�li�i
	\param height G�r�nt�n�n piksel olarak y�ksekli�i

	\ref gpuInvert kernelini Grid ve Block boyutlar�n� ayarlayarak �a��ran metod.
*/
void deviceInvertLaunch(
	float *d_Image,
	int width,
	int height
	)
{
	 // launch kernel
	dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );

#if ENABLE_TIMING_CODE

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#endif

    gpuInvert<<< dimGrid, dimBlock >>>( d_Image, width, height);

#if ENABLE_TIMING_CODE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

    // block until the device has completed
    hipDeviceSynchronize();
	
	printf("gpuInvert kernel time: %.3f ms\n", elapsedTime);
#endif

	hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");
}
